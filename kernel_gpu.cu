
#include <hip/hip_runtime.h>
__global__ void grayscale(float4* imagem, int width, int height)
{
	const int i = blockIdx.x * (blockDim.x * blockDim.y) + blockDim.x * threadIdx.y + threadIdx.x;

	if(i < width * height)
	{
		float v = 0.3 * imagem[i].x + 0.6 * imagem[i].y + 0.1 * imagem[i].z;
		imagem[i] = make_float4(v, v, v, 0);
	}
}


extern "C" void cuda_grayscale(float* imagem, int width, int height, dim3 blocks, dim3 block_size)
{
	grayscale <<< blocks, block_size >>> ((float4*)imagem, width, height);
}

